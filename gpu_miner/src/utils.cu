#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "utils.h"
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA sprintf alternative for nonce finding. Converts integer to its string representation. Returns string's length.
__device__ int intToString(uint64_t num, char* out) {
    if (num == 0) {
        out[0] = '0';
        out[1] = '\0';
        return 1;
    }

    int i = 0;
    while (num != 0) {
        int digit = num % 10;
        num /= 10;
        out[i++] = '0' + digit;
    }

    // Reverse the string
    for (int j = 0; j < i / 2; j++) {
        char temp = out[j];
        out[j] = out[i - j - 1];
        out[i - j - 1] = temp;
    }
    out[i] = '\0';
    return i;
}

// CUDA strlen implementation.
__host__ __device__ size_t d_strlen(const char *str) {
    size_t len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

// CUDA strcpy implementation.
__device__ void d_strcpy(char *dest, const char *src) {
    int i = 0;
    while ((dest[i] = src[i]) != '\0') {
        i++;
    }
}

// CUDA strcat implementation.
__device__ void d_strcat(char *dest, const char *src) {
    while (*dest != '\0') {
        dest++;
    }
    while (*src != '\0') {
        *dest = *src;
        dest++;
        src++;
    }
    *dest = '\0';
}

// Compute SHA256 and convert to hex
__host__ __device__ void apply_sha256(const BYTE *input, BYTE *output) {
    size_t input_length = d_strlen((const char *)input);
    SHA256_CTX ctx;
    BYTE buf[SHA256_BLOCK_SIZE];
    const char hex_chars[] = "0123456789abcdef";

    sha256_init(&ctx);
    sha256_update(&ctx, input, input_length);
    sha256_final(&ctx, buf);

    for (size_t i = 0; i < SHA256_BLOCK_SIZE; i++) {
        output[i * 2] = hex_chars[(buf[i] >> 4) & 0x0F];  // High nibble
        output[i * 2 + 1] = hex_chars[buf[i] & 0x0F];     // Low nibble
    }
    output[SHA256_BLOCK_SIZE * 2] = '\0'; // Null-terminate
}

// Compare two hashes
__host__ __device__ int compare_hashes(BYTE* hash1, BYTE* hash2) {
    for (int i = 0; i < SHA256_HASH_SIZE; i++) {
        if (hash1[i] < hash2[i]) {
            return -1; // hash1 is lower
        } else if (hash1[i] > hash2[i]) {
            return 1; // hash2 is lower
        }
    }
    return 0; // hashes are equal
}

// Kernel for computing transaction hashes
__global__ void compute_transaction_hashes_kernel(BYTE *transactions, BYTE *hashes, int transaction_size, int n) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        apply_sha256(transactions + idx * transaction_size, hashes + idx * SHA256_HASH_SIZE);
    }
}

// Kernel for constructing one level of the Merkle tree
__global__ void construct_merkle_level_kernel(BYTE *hashes, BYTE *next_level_hashes, int n) {
    __shared__ BYTE shared_hashes[256 * SHA256_HASH_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Încarcă hash-urile în shared memory
    if (idx < n) {
        // Copiază hash-ul din memoria globală în shared memory
        for (int i = 0; i < SHA256_HASH_SIZE; i++) {
            shared_hashes[tid * SHA256_HASH_SIZE + i] = hashes[idx * SHA256_HASH_SIZE + i];
        }
    }
    __syncthreads();

    // Calculează hash-urile pentru nivelul următor
    // Doar thread-urile care pot forma perechi valide
    if (tid < blockDim.x / 2 && blockIdx.x * blockDim.x / 2 + tid < (n + 1) / 2) {
        BYTE combined[2 * SHA256_HASH_SIZE];
        
        // Indexi în shared memory
        int local_idx1 = tid * 2;
        int local_idx2 = tid * 2 + 1;
        
        // Indexul global pentru primul hash din pereche
        int global_idx1 = blockIdx.x * blockDim.x + local_idx1;
        
        // Copiază primul hash
        d_strcpy((char*)combined, (const char*)&shared_hashes[local_idx1 * SHA256_HASH_SIZE]);
        
        // Verifică dacă al doilea hash există
        if (global_idx1 + 1 < n && local_idx2 < blockDim.x) {
            d_strcat((char*)combined, (const char*)&shared_hashes[local_idx2 * SHA256_HASH_SIZE]);
        } else {
            // Duplică primul hash dacă al doilea nu există
            d_strcat((char*)combined, (const char*)&shared_hashes[local_idx1 * SHA256_HASH_SIZE]);
        }
        
        // Calculează indexul pentru scrierea rezultatului în memoria globală
        int output_idx = blockIdx.x * blockDim.x / 2 + tid;
        
        // Calculează și scrie hash-ul combinat
        apply_sha256(combined, next_level_hashes + output_idx * SHA256_HASH_SIZE);
    }

}   

// CUDA implementation for constructing the Merkle root
void construct_merkle_root(int transaction_size, BYTE *transactions, int max_transactions_in_a_block, int n, BYTE merkle_root[SHA256_HASH_SIZE]) {
    // Handle edge cases
    if (n == 0) {
        memset(merkle_root, 0, SHA256_HASH_SIZE);
        return;
    }
    
    if (n == 1) {
        apply_sha256(transactions, merkle_root);
        return;
    }
    
    // Check for hipMalloc success
    BYTE *d_transactions;
    // Check if device has enough memory
    hipMalloc((void **) &d_transactions, n * transaction_size);
    hipMemcpy(d_transactions, transactions, n * transaction_size, hipMemcpyHostToDevice);
    
    // Device memory for current level hashes
    BYTE *d_current_hashes;
    hipMalloc(&d_current_hashes, n * SHA256_HASH_SIZE);
    
    // Compute initial transaction hashes
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    compute_transaction_hashes_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_transactions, d_current_hashes, transaction_size, n
    );
    hipDeviceSynchronize();
 
    // Free transaction memory as it's no longer needed
    hipFree(d_transactions);
    
    // Construct the Merkle root
    int current_level_size = n;
    while (current_level_size > 1) {
        int next_level_size = (current_level_size + 1) / 2;
        BYTE *d_next_hashes;
        hipMalloc(&d_next_hashes, next_level_size * SHA256_HASH_SIZE);
        // Compute the needed number of blocks
        int no_of_blocks = (current_level_size / threadsPerBlock);
        if (current_level_size % threadsPerBlock != 0) {
            no_of_blocks++;
        }

        // Launch kernel to construct next level of hashes
        construct_merkle_level_kernel<<<no_of_blocks, threadsPerBlock>>>(
            d_current_hashes, d_next_hashes, current_level_size
        );
        hipDeviceSynchronize();

        hipFree(d_current_hashes);
        // Allocate memory for the current level hashes
        hipMalloc((void **)&d_current_hashes, next_level_size * SHA256_HASH_SIZE);
        // Copy the next level hashes to the current level
        hipMemcpy(d_current_hashes, d_next_hashes, next_level_size * SHA256_HASH_SIZE, hipMemcpyDeviceToDevice);
        hipFree(d_next_hashes);
        current_level_size = next_level_size;
    }

    // Copy the final Merkle root to the host
    hipMemcpy(merkle_root, d_current_hashes, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    hipFree(d_current_hashes);
}


// CUDA implementation for finding a valid nonce
int find_nonce(BYTE *difficulty, uint32_t max_nonce, BYTE *block_content, size_t current_length, BYTE *block_hash, uint32_t *valid_nonce) {
    char nonce_string[NONCE_SIZE];

    for (uint32_t nonce = 0; nonce <= max_nonce; nonce++) {
        sprintf(nonce_string, "%u", nonce);
        strcpy((char *)block_content + current_length, nonce_string);
        apply_sha256(block_content, block_hash);

        if (compare_hashes(block_hash, difficulty) <= 0) {
            *valid_nonce = nonce;
            return 0;
        }
    }

    return 1;
}


__global__ void dummy_kernel() {}

// Warm-up function for the GPU
void warm_up_gpu() {
    BYTE *dummy_data;
    hipMalloc((void **)&dummy_data, 256);
    dummy_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    hipFree(dummy_data);
}