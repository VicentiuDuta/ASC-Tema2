#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "utils.h"
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// CUDA sprintf alternative for nonce finding. Converts integer to its string representation. Returns string's length.
__device__ int intToString(uint64_t num, char* out) {
    if (num == 0) {
        out[0] = '0';
        out[1] = '\0';
        return 2;
    }

    int i = 0;
    while (num != 0) {
        int digit = num % 10;
        num /= 10;
        out[i++] = '0' + digit;
    }

    // Reverse the string
    for (int j = 0; j < i / 2; j++) {
        char temp = out[j];
        out[j] = out[i - j - 1];
        out[i - j - 1] = temp;
    }
    out[i] = '\0';
    return i;
}

// CUDA strlen implementation.
__host__ __device__ size_t d_strlen(const char *str) {
    size_t len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

// CUDA strcpy implementation.
__device__ void d_strcpy(char *dest, const char *src){
    int i = 0;
    while ((dest[i] = src[i]) != '\0') {
        i++;
    }
}

// CUDA strcat implementation.
__device__ void d_strcat(char *dest, const char *src){
    while (*dest != '\0') {
        dest++;
    }
    while (*src != '\0') {
        *dest = *src;
        dest++;
        src++;
    }
    *dest = '\0';
}

// Compute SHA256 and convert to hex
__host__ __device__ void apply_sha256(const BYTE *input, BYTE *output) {
    size_t input_length = d_strlen((const char *)input);
    SHA256_CTX ctx;
    BYTE buf[SHA256_BLOCK_SIZE];
    const char hex_chars[] = "0123456789abcdef";

    sha256_init(&ctx);
    sha256_update(&ctx, input, input_length);
    sha256_final(&ctx, buf);

    for (size_t i = 0; i < SHA256_BLOCK_SIZE; i++) {
        output[i * 2]     = hex_chars[(buf[i] >> 4) & 0x0F];  // High nibble
        output[i * 2 + 1] = hex_chars[buf[i] & 0x0F];         // Low nibble
    }
    output[SHA256_BLOCK_SIZE * 2] = '\0'; // Null-terminate
}

// Compare two hashes
__host__ __device__ int compare_hashes(BYTE* hash1, BYTE* hash2) {
    for (int i = 0; i < SHA256_HASH_SIZE; i++) {
        if (hash1[i] < hash2[i]) {
            return -1; // hash1 is lower
        } else if (hash1[i] > hash2[i]) {
            return 1; // hash2 is lower
        }
    }
    return 0; // hashes are equal
}
__global__ void compute_transaction_hashes_kernel(BYTE *transactions, BYTE *hashes, int transaction_size, int n) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        apply_sha256(transactions + idx * transaction_size, hashes + idx * SHA256_HASH_SIZE);
    }
}

__global__ void construct_merkle_level_kernel(BYTE *hashes, int n, BYTE *device_next_hashes) {
        // Allocate shared memory for the current level
        __shared__ BYTE shared_hashes[256 * SHA256_HASH_SIZE];
        unsigned int tid = threadIdx.x;
        unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

        // Load the hashes into shared memory 
        if (idx < n) {
            for (int i = 0; i < SHA256_HASH_SIZE; i++) {
                shared_hashes[tid * SHA256_HASH_SIZE + i] = hashes[idx * SHA256_HASH_SIZE + i];
            }
        } 
        
        // Synchronize threads to ensure all data is loaded
        __syncthreads();
        
        // Compute the next level of hashes only with the first half of threads
        if (idx < n / 2) {
            BYTE combined_hash[2 * SHA256_HASH_SIZE];
            int idx1 = tid * 2;
            int idx2 = tid * 2 + 1;

            // Copy the first hash
            d_strcpy((char *) combined_hash, (const char *) (shared_hashes + idx1 * SHA256_HASH_SIZE));
            // If the second hash exists, concatenate it
            if (idx2 < blockDim.x && idx * 2 + 1 < n) {
                d_strcat((char *) combined_hash, (const char *) (shared_hashes + idx2 * SHA256_HASH_SIZE));
            } else {
                // If the second hash does not exist, duplicate the first hash
                d_strcat((char *) combined_hash, (const char *) (shared_hashes + idx1 * SHA256_HASH_SIZE));
            }
            
            // Compute the hash of the combined hashes
            apply_sha256((BYTE *) combined_hash, device_next_hashes + idx * SHA256_HASH_SIZE);
        }
} 

// TODO 1: Implement this function in CUDA
void construct_merkle_root(int transaction_size, BYTE *transactions, int max_transactions_in_a_block, int n, BYTE merkle_root[SHA256_HASH_SIZE]) {
    BYTE (*hashes)[SHA256_HASH_SIZE] = (BYTE (*)[SHA256_HASH_SIZE])malloc(max_transactions_in_a_block * SHA256_HASH_SIZE);
    // Allocate device memory for transactions
    BYTE *device_transactions;
    hipMalloc((void **) &device_transactions, n * transaction_size);

    // Allocate device memory for hashes
    BYTE *device_hashes;
    hipMalloc((void **) &device_hashes, n * SHA256_HASH_SIZE);

    // Copy transactions to device
    hipMemcpy(device_transactions, transactions, n * transaction_size, hipMemcpyHostToDevice);


    // Declare kernel parameters
    const size_t block_size = 256;
    size_t num_blocks = n / block_size;
    if (n % block_size != 0) {
        num_blocks++;
    }

    // Launch kernel to compute transaction hashes
    compute_transaction_hashes_kernel<<<num_blocks, block_size>>>(device_transactions, device_hashes, transaction_size, n);
    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Free device memory
    hipFree(device_transactions);
    
    // Allocate device memory for next level hashes
    BYTE *device_next_hashes;
    hipMalloc((void **) &device_next_hashes, (n / 2 + n % 2) * SHA256_HASH_SIZE);

    while (n > 1) {
        int new_n = n / 2;
        if (n % 2 != 0)
            new_n++; // if odd, duplicate last hash

        num_blocks = n / block_size;
        if(n % block_size != 0) {
            num_blocks++; // round up
        }

        construct_merkle_level_kernel<<<num_blocks, block_size>>>(device_hashes, n, device_next_hashes);
        hipDeviceSynchronize();

        // Copy the next level hashes to the device
        BYTE *temp = device_hashes;
        device_hashes = device_next_hashes;
        device_next_hashes = temp;
        n = new_n;
    }

    // Copy the final merkle root to host
    hipMemcpy(merkle_root, device_hashes, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_hashes);
    hipFree(device_next_hashes);
    // Free host memory
    free(hashes);

}

void find_nonce_kernel(BYTE *difficulty, uint32_t max_nonce, BYTE *block_content, size_t current_length, BYTE *block_hash, uint32_t *valid_nonce, uint32_t *found_nonce_flag) {

}

// TODO 2: Implement this function in CUDA
int find_nonce(BYTE *difficulty, uint32_t max_nonce, BYTE *block_content, size_t current_length, BYTE *block_hash, uint32_t *valid_nonce) {
    // Allocate device memory for block content
    BYTE *device_difficulty;
    hipMalloc((void **) &device_difficulty, SHA256_HASH_SIZE);
    hipMemcpy(device_difficulty, difficulty, SHA256_HASH_SIZE, hipMemcpyHostToDevice);

    BYTE *device_block_content;
    hipMalloc((void **) &device_block_content, current_length);
    hipMemcpy(device_block_content, block_content, current_length, hipMemcpyHostToDevice);

    BYTE *device_block_hash;
    hipMalloc((void **) &device_block_hash, SHA256_HASH_SIZE);
    hipMemcpy(device_block_hash, block_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);

    uint32_t *device_valid_nonce;
    hipMalloc((void **) &device_valid_nonce, sizeof(uint32_t));

    int* device_found_nonce;
    hipMalloc((void **) &device_found_nonce, sizeof(int));
    hipMemset(device_found_nonce, 0, sizeof(int));

    // Declare kernel parameters
    const size_t block_size = 512;
    size_t num_blocks = max_nonce / block_size;
    if (max_nonce % block_size != 0) {
        num_blocks++;
    }
}


__global__ void dummy_kernel() {}

// Warm-up function
void warm_up_gpu() {
    BYTE *dummy_data;
    hipMalloc((void **)&dummy_data, 256);
    dummy_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    hipFree(dummy_data);
}
